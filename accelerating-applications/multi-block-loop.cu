// Run Command
//!nvcc -o multi-block-loop 04-loops/02-multi-block-loop.cu -run


#include <hip/hip_runtime.h>
#include <stdio.h>

/*
 * Refactor `loop` to be a CUDA Kernel. The new kernel should
 * only do the work of 1 iteration of the original loop.
 */

__global__ void loop()
{
  /*for (int i = 0; i < N; ++i)
  {
    printf("This is iteration number %d\n", i);
  }*/

  printf("This is iteration number %d\n", ((blockIdx.x * blockDim.x) + threadIdx.x));
}

int main()
{
  /*
   * When refactoring `loop` to launch as a kernel, be sure
   * to use the execution configuration to control how many
   * "iterations" to perform.
   *
   * For this exercise, only use 1 block of threads.
   */

  int N = 10;
  loop<<<2, N/2>>>();

  hipDeviceSynchronize();
}
