// Run Command
// !nvcc -o vector-add 07-vector-add/01-vector-add.cu -run


#include <hip/hip_runtime.h>
#include <stdio.h>

void initWith(float num, float *a, int N)
{
  for(int i = 0; i < N; i++) 
  { 
    a[i] = num;
  }
}

__global__ void addVectorsInto(float *result, float *a, float *b, int N)
{
  int i;
  i = blockIdx.x * blockDim.x + threadIdx.x;
  while(i < N) {
    result[i] = a[i] + b[i];
    i += (blockDim.x * gridDim.x);
  }
}

void checkElementsAre(float target, float *array, int N)
{
  for(int i = 0; i < N; i++)
  {
    if(array[i] != target)
    {
      printf("FAIL: array[%d] - %0.0f does not equal %0.0f\n", i, array[i], target);
      exit(1);
    }
  }
  printf("SUCCESS! All values added correctly.\n");
}

int main()
{
  const int N = 2<<20;
  size_t size = N * sizeof(float);

  float *a;
  float *b;
  float *c;

  hipMallocManaged(&a, size);
  hipMallocManaged(&b, size);
  hipMallocManaged(&c, size);

  //a = (float *)malloc(size);
  //b = (float *)malloc(size);
  //c = (float *)malloc(size);

  initWith(3, a, N);
  initWith(4, b, N);
  initWith(0, c, N);

  addVectorsInto<<<32, 256>>>(c, a, b, N);
  hipDeviceSynchronize();

  checkElementsAre(7, c, N);

  hipFree(a);
  hipFree(b);
  hipFree(c);
}
