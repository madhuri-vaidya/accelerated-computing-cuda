// Run command
// nvcc -o hello-gpu 01-hello/01-hello-gpu.cu -run


#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>
#include <ctime>
#include <stdio.h>


using namespace std;

void helloCPU() { cout << "Hello from the CPU" << endl; }

/*
* Refactor the `helloGPU` definition to be a kernel that can be launched on the GPU. 
* Update its message to read "Hello from the GPU!"
*/

__global__ void helloGPU() { printf("Hello from the GPU\n"); }


int main()
{
    /*
    * Refactor this call to `helloGPU` so that it launches as a kernel on the GPU.
    */
    auto start = chrono::system_clock::now();
    helloGPU<<<1, 1>>>();

    /*
    * Add code below to synchronize on the completion of the `helloGPU` kernel completion before continuing the CPU thread.
    */

    hipDeviceSynchronize();

    auto gpu1 = chrono::system_clock::now();
    chrono::duration<double> gpu_duration_1 = gpu1 - start;
    cout << "Time taken for GPU function is " << gpu_duration_1.count() << endl;

    helloCPU();

    auto cpu = chrono::system_clock::now();
    chrono::duration<double> cpu_duration = cpu - gpu1;
    cout << "Time taken for CPU function is " << cpu_duration.count() << endl;

    helloGPU<<<1, 1>>>();
    hipDeviceSynchronize();

    auto gpu2 = chrono::system_clock::now();
    chrono::duration<double> gpu_duration_2 = gpu2 - cpu;
    cout << "Time taken for GPU function is " << gpu_duration_2.count() << endl;

    return 0;
}
