//Run Command
// !nvcc -o single-block-loop 04-loops/01-single-block-loop.cu -run


#include <hip/hip_runtime.h>
#include <stdio.h>

/*
 * Refactor `loop` to be a CUDA Kernel. The new kernel should
 * only do the work of 1 iteration of the original loop.
 */

__global__ void loop()
{
  /*for (int i = 0; i < N; ++i)
  {
    printf("This is iteration number %d\n", i);
  }*/

  printf("This is iteration number %d\n", threadIdx.x);
}

int main()
{
  /*
   * When refactoring `loop` to launch as a kernel, be sure
   * to use the execution configuration to control how many
   * "iterations" to perform.
   *
   * For this exercise, only use 1 block of threads.
   */

  int N = 10;
  loop<<<1, N>>>();

  hipDeviceSynchronize();
}