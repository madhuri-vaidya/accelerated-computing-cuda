// Run Command
// !nvcc -o add-error-handling 06-errors/01-add-error-handling.cu -run


#include <hip/hip_runtime.h>
#include <stdio.h>

void init(int *a, int N)
{
  int i;
  for (i = 0; i < N; ++i)
  {
    a[i] = i;
  }
}

__global__ void doubleElements(int *a, int N)
{

  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = gridDim.x * blockDim.x;

  for (int i = idx; i < N + stride; i += stride)
  {
    a[i] *= 2;
  }
}

bool checkElementsAreDoubled(int *a, int N)
{
  int i;
  for (i = 0; i < N; ++i)
  {
    if (a[i] != i*2) return false;
  }
  return true;
}

int main()
{
  /*
   * Add error handling to this source code to learn what errors
   * exist, and then correct them. Googling error messages may be
   * of service if actions for resolving them are not clear to you.
   */

  int N = 10000;
  int *a;
  hipError_t err_malloc, err_last, err_sync;

  size_t size = N * sizeof(int);
  err_malloc = hipMallocManaged(&a, size);
  if (err_malloc != hipSuccess) { printf("Error: %s\n", hipGetErrorString(err_malloc)); }
 
  init(a, N);

  size_t threads_per_block = 1024;
  size_t number_of_blocks = 32;

  doubleElements<<<number_of_blocks, threads_per_block>>>(a, N);
  err_last = hipGetLastError();
  if (err_last != hipSuccess) { printf("Error: %s\n", hipGetErrorString(err_last)); }
  
  
  err_sync = hipDeviceSynchronize();
  if (err_sync != hipSuccess) { printf("Error: %s\n", hipGetErrorString(err_sync)); }

  bool areDoubled = checkElementsAreDoubled(a, N);
  printf("All elements were doubled? %s\n", areDoubled ? "TRUE" : "FALSE");

  hipFree(a);
}
